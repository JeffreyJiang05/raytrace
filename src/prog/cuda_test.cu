#include <cstdio>

#include "gpu/gpu.hpp"
#include "math/functions.hpp"
#include "math/linalg/vec.hpp"
#include "math/sampling.hpp"

KERNEL test_vectors()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    math::vector<int, 2> v0{ index, index + 1 }, v1{ index - 1, index - 2 };
    auto res = v0 + v1;
    printf("%d - [%d %d]\n", index, res[0], res[1]);
}

int main()
{
    test_vectors<<<4, 4>>>();
    hipDeviceSynchronize();
    return 0;
}